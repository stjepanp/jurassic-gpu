#include "sum_vector.h"

#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

#include "gpu_summation.h"

__host__ void add_vector(vector *a, const vector *b) {
  assert(a->n == b->n);
  int *a_gpu, *b_gpu;
  hipMalloc(&a_gpu, a->n * sizeof(int));
  hipMalloc(&b_gpu, b->n * sizeof(int));
  hipMemcpy(a_gpu, a->array, a->n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b_gpu, b->array, b->n * sizeof(int), hipMemcpyHostToDevice); 
  dim3 blockDim(64, 1, 1);
  dim3 gridDim(a->n / 64 + 1);
  sum <<<gridDim, blockDim>>> (a_gpu, b_gpu, a->n);
  hipError_t err = hipGetLastError();
  if(err) {
    printf("Error!\n");
    exit(0);
  }
  hipMemcpy(a->array, a_gpu, a->n * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(a_gpu);
  hipFree(b_gpu);
}
